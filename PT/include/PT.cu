#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gtest/gtest.h"

template <typename... PartitionsT>
__device__ inline static void pair_example(
    std::pair<PartitionsT, int>... partitions)
{
    for (auto& p : {partitions...}) {
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            printf("\n p.first= %d", p.first);
            printf("\n p.second= %d", p.second);
        }
    }
}


template <typename... PartitionsT>
__device__ inline static void tuple_example(
    std::tuple<PartitionsT, int>... partitions)
{
    for (auto& p : {partitions...}) {
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            printf("\n p<0>= %d", std::get<0>(p));
            printf("\n p<1>= %d", std::get<1>(p));
        }
    }
}

__global__ void exec_kernel()
{
    int i = 1;
    int j = 2;

    pair_example(std::make_pair(i, j), std::make_pair(j, i));

    tuple_example(std::make_tuple(i, j), std::make_tuple(j, i));
}

TEST(Test, exe)
{
    exec_kernel<<<1, 1>>>();
    auto err = hipDeviceSynchronize();
    EXPECT_EQ(err, hipSuccess);
}

int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
